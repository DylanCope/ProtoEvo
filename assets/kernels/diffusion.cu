
#include <hip/hip_runtime.h>
__device__ const int FILTER_SIZE = 3;

extern "C"
__global__ void kernel(
    unsigned int width,
    unsigned int height,
    unsigned int channels,
    unsigned char *img,
    unsigned char *result)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // See voidStartDistance in SimulationSettings
    float world_radius = 30.0;

    float cellSizeX = 2 * world_radius / ((float) width);
    float cellSizeY = 2 * world_radius / ((float) height);
    float world_x = -world_radius + cellSizeX * x;
    float world_y = -world_radius + cellSizeY * y;
    float dist2_to_world_centre = world_x*world_x + world_y*world_y;

    // set alpha decay to zero as we approach the void
    float decay = 0.0;

    float void_p = 0.9;
    if (dist2_to_world_centre > void_p * void_p * world_radius * world_radius) {
        float dist_to_world_centre = sqrtf(dist2_to_world_centre);
        // lerp from 1.0 to 0.0 for distance between void_p*world_radius and world_radius
        decay = 0.9995 * (1.0 - (dist_to_world_centre - void_p * world_radius) / ((1.0 - void_p) * world_radius));
        if (decay < 0.0) {
            decay = 0.0;
        }
    } else {
        decay = 0.995;
    }

    int alpha_channel = channels - 1;

    float final_alpha = 0.0;
    int radius = (FILTER_SIZE - 1) / 2;
    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            int x_ = x + i;
            int y_ = y + j;
            if (x_ < 0 || x_ >= width || y_ < 0 || y_ >= height) {
                continue;
            }
            float val = img[(y_*width + x_)*channels + alpha_channel];
            final_alpha += val / 255.0f;
        }
    }
    final_alpha = decay * final_alpha / ((float) (FILTER_SIZE*FILTER_SIZE));
    result[(y*width + x)*channels + alpha_channel] = (unsigned char) (255 * final_alpha);

    if (final_alpha < 5.0 / 255.0) {
        for (int i = 0; i < channels - 1; i++) {
            result[(y*width + x)*channels + i] = 0;
        }
    }

    float final_value = 0.0;
    // assume that the last channel is alpha
    for (int c = 0; c < channels - 1; c++) {
        final_value = 0.0;
        for (int i = -radius; i <= radius; i++) {
            for (int j = -radius; j <= radius; j++) {
                int x_ = x + i;
                int y_ = y + j;
                if (x_ < 0 || x_ >= width || y_ < 0 || y_ >= height) {
                    continue;
                }
                float alpha = decay * ((float) (img[(y_*width + x_)*channels + alpha_channel])) / 255.0;
                float val = ((float) img[(y_*width + x_)*channels + c]) / 255.0;
                final_value += val * alpha;
            }
        }
        final_value = final_value / ((float) (FILTER_SIZE*FILTER_SIZE));
        final_value = decay * 255 * final_value / final_alpha;

        result[(y*width + x)*channels + c] = (unsigned char) (final_value);
    }
}