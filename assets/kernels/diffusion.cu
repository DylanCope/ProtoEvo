
#include <hip/hip_runtime.h>
__device__ const int FILTER_SIZE = 5;

extern "C"
__global__ void kernel(
    unsigned int width,
    unsigned int height,
    unsigned int channels,
    unsigned int *img,
    unsigned int *result)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float decay = 0.9995;
    int alpha_channel = channels - 1;

    float final_alpha = 0.0;
    int radius = (FILTER_SIZE - 1) / 2;
    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            int x_ = x + i;
            int y_ = y + j;
            if (x_ < 0 || x_ >= width || y_ < 0 || y_ >= height) {
                continue;
            }
            final_alpha += ((float) img[(y_*width + x_)*channels + alpha_channel]) / 255.0;
        }
    }
    final_alpha = decay * final_alpha / ((float) (FILTER_SIZE*FILTER_SIZE));
    result[(y*width + x)*channels + alpha_channel] = (int) (255 * final_alpha);

    float final_value = 0.0;
    // assume that the last channel is alpha
    for (int c = 0; c < channels - 1; c++) {
        final_value = 0.0;
        for (int i = -radius; i <= radius; i++) {
            for (int j = -radius; j <= radius; j++) {
                int x_ = x + i;
                int y_ = y + j;
                if (x_ < 0 || x_ >= width || y_ < 0 || y_ >= height) {
                    continue;
                }
                float alpha = decay * ((float) (img[(y_*width + x_)*channels + alpha_channel])) / 255.0;
                float val = ((float) img[(y_*width + x_)*channels + c]) / 255.0;
                final_value += val * alpha;
            }
        }
        final_value = final_value / ((float) (FILTER_SIZE*FILTER_SIZE));
        final_value = decay * 255 * final_value / final_alpha;
        result[(y*width + x)*channels + c] = (int) (final_value);
    }
}