
#include <hip/hip_runtime.h>
__device__ const int FILTER_SIZE = 3;

extern "C"
__global__ void kernel(
    unsigned int width,
    unsigned int height,
    unsigned int *img,
    unsigned int *filter,
    unsigned int *result)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int sum = 0;
    sum = 0;
    for(int filterY=0; filterY < FILTER_SIZE; filterY++) {
        for(int filterX=0; filterX < FILTER_SIZE; filterX++) {
            sum += img[ ((y + filterY) * width) + x + filterX ] * filter[ (filterY * FILTER_SIZE) + filterX ];
        }
    }

    if(y + 1 < height && x + 1 < width) {
        result[((y + 1) * width) + x + 1] = sum / 16;
    }
}